
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <chrono>
#include <iostream>

using namespace std::chrono;

// Error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#define N 500000

// Simple short kernels
__global__
void kernel_a(float* x, float* y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) y[idx] += 1;
}

__global__
void kernel_c(float* x, float* y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) y[idx] += 1;
}

int main() {

    hipStream_t stream1;
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    hipblasSetStream(cublas_handle, stream1);

    // Set up host data and initialize
    float* h_x = (float*) malloc(N * sizeof(float));
    float* h_y = (float*) malloc(N * sizeof(float));
    for (int i = 0; i < N; ++i) {
        h_x[i] = float(i);
        h_y[i] = float(i);
    }

    // Print out the first 25 values of h_y
    for (int i = 0; i < 25; ++i) {
        printf("%2.0f ", h_y[i]);
    }
    printf("\n");

    // Set up device data
    float* d_x;
    float* d_y;
    float d_a = 5.0;
    hipMalloc((void**) &d_x, N * sizeof(float));
    hipMalloc((void**) &d_y, N * sizeof(float));
    cudaCheckErrors("hipMalloc failed");

    hipblasSetVector(N, sizeof(h_x[0]), h_x, 1, d_x, 1);
    hipblasSetVector(N, sizeof(h_y[0]), h_y, 1, d_y, 1);
    cudaCheckErrors("hipblasSetVector failed");

    // Set up graph
    hipGraph_t graph;
    hipGraph_t libraryGraph;
    std::vector<hipGraphNode_t> nodeDependencies;
    hipGraphNode_t kernelNode1, kernelNode2, libraryNode;

    hipKernelNodeParams kernelNode1Params {0};
    hipKernelNodeParams kernelNode2Params {0};

    hipGraphCreate(&graph, 0);
    cudaCheckErrors("hipGraphCreate failure");

    // kernel_a and kernel_c use same args
    void *kernelArgs[2] = {(void *)&d_x, (void *)&d_y};

    int threads = 512;
    int blocks = (N + (threads - 1) / threads);

    // Adding 1st node, kernel_a
    kernelNode1Params.func = (void *)kernel_a;
    kernelNode1Params.gridDim = dim3(blocks, 1, 1);
    kernelNode1Params.blockDim = dim3(threads, 1, 1);
    kernelNode1Params.sharedMemBytes = 0;
    kernelNode1Params.kernelParams = (void **)kernelArgs;
    kernelNode1Params.extra = NULL;

    hipGraphAddKernelNode(&kernelNode1, graph, NULL, 0, &kernelNode1Params);
    cudaCheckErrors("Adding kernelNode1 failed");

    nodeDependencies.push_back(kernelNode1);

    // Timing cublasSaxpy
    auto start = high_resolution_clock::now();

    // Adding 2nd node, libraryNode, with kernelNode1 as dependency
    hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);
    cudaCheckErrors("Stream capture begin failure");

    // Library call
    hipblasSaxpy(cublas_handle, N, &d_a, d_x, 1, d_y, 1);
    cudaCheckErrors("hipblasSaxpy failure");

    hipStreamEndCapture(stream1, &libraryGraph);
    cudaCheckErrors("Stream capture end failure");

    auto end = high_resolution_clock::now();
    duration<double> saxpy_time = duration_cast<duration<double>>(end - start);
    std::cout << "hipblasSaxpy Execution Time: " << saxpy_time.count() << " s" << std::endl;

    hipGraphAddChildGraphNode(&libraryNode, graph, nodeDependencies.data(), nodeDependencies.size(), libraryGraph);
    cudaCheckErrors("Adding libraryNode failed");

    nodeDependencies.clear();
    nodeDependencies.push_back(libraryNode);

    // Adding 3rd node, kernel_c
    kernelNode2Params.func = (void *)kernel_c;
    kernelNode2Params.gridDim = dim3(blocks, 1, 1);
    kernelNode2Params.blockDim = dim3(threads, 1, 1);
    kernelNode2Params.sharedMemBytes = 0;
    kernelNode2Params.kernelParams = (void **)kernelArgs;
    kernelNode2Params.extra = NULL;

    hipGraphAddKernelNode(&kernelNode2, graph, nodeDependencies.data(), nodeDependencies.size(), &kernelNode2Params);
    cudaCheckErrors("Adding kernelNode2 failed");

    nodeDependencies.clear();
    nodeDependencies.push_back(kernelNode2);

    hipGraphNode_t *nodes = NULL;
    size_t numNodes = 0;
    hipGraphGetNodes(graph, nodes, &numNodes);
    cudaCheckErrors("Graph get nodes failed");
    printf("Number of the nodes in the graph = %zu\n", numNodes);

    hipGraphExec_t instance;
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
    cudaCheckErrors("Graph instantiation failed");

    // Timing the graph launch
    start = high_resolution_clock::now();

    // Launch the graph instance 100 times
    for (int i = 0; i < 100; ++i) {
        hipGraphLaunch(instance, stream1);
        hipStreamSynchronize(stream1);
    }
    cudaCheckErrors("Graph launch failed");

    end = high_resolution_clock::now();
    duration<double> graph_launch_time = duration_cast<duration<double>>(end - start);
    std::cout << "Graph Launch Execution Time (100 launches): " << graph_launch_time.count() << " s" << std::endl;

    hipDeviceSynchronize();

    // Copy memory back to host
    hipMemcpy(h_y, d_y, N, hipMemcpyDeviceToHost);
    cudaCheckErrors("Finishing memcpy failed");

    hipDeviceSynchronize();

    // Print out the first 25 values of h_y
    for (int i = 0; i < 25; ++i) {
        printf("%2.0f ", h_y[i]);
    }
    printf("\n");

    // Clean up
    free(h_x);
    free(h_y);
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(cublas_handle);
    hipStreamDestroy(stream1);
    hipGraphDestroy(graph);
    hipGraphDestroy(libraryGraph);

    return 0;
}
